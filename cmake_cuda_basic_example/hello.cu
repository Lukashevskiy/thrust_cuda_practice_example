#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <vector>
__global__ void test_kernel(int *p1, int *p2, int length) {
    for(int i = 0; i < length; ++i){
        p2[i] = p1[i] + p2[i];
    }
}

int main(){

    std::random_device rd;
    std::mt19937 gen{rd()}; 
    std::uniform_int_distribution distrib{1, 100000};

    std::vector v1, v2;
    v1.reserve(100000);
    v2.reserve(100000);
    std::generate(v1.begin(), v1.end(), [=]{ return distrib(gen); });
    std::generate(v2.begin(), v2.end(), [=]{ return distrib(gen); });
    
    int *d1, *d2;
    size_t s = v1.size() * sizeof(int);

    hipMalloc(reinterpret_cast<void**>(&d1), s);
    hipMalloc(reinterpret_cast<void**>(&d2), s);
    
    hipMemcpy(d1, &v1[0], s, hipMemcpyHostToDevice);
    hipMemcpy(d2, &v2[0], s, hipMemcpyHostToDevice);

    for(auto el: v1){
        std:: cout << el << " ";
    }
    std::cout << std::endl;

    for(auto el: v2){
        std:: cout << el << " ";
    }
    std::cout << std::endl;

    test_kernel<<<1, 1>>>(d1, d2, v1.size());

    hipMemcpy(&v2[0], d2, s, hipMemcpyDeviceToHost);

    for(auto el: v2){
        std:: cout << el << " ";
    }

    std::cout << std::endl;

    hipFree(d1);
    hipFree(d2);

    std::cout << "Hello, world!" << std::endl;
    
    return 0;
}